#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_hello(){
	int a=99;
    	printf("Hello World from GPU!\n");
    	printf("a=%d \n",a);
}

int main() {
   cuda_hello<<<1,1>>>();
	printf("Hello World from CPU!\n"); 
 hipDeviceSynchronize();
    return 0;
}
