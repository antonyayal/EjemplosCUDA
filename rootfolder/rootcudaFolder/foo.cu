#include "hip/hip_runtime.h"
/*
 * nvcc *.c -o foo
 * ./foo
 * 
 */

#include <stdio.h>
#include "hellofunFolder/hellofun.h"

__global__ void foo() {
    printf("Hola desde tarjeta\n");
}

int main(){
    
  foo<<<1,1>>>();

  hipDeviceSynchronize();
  //printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
    
helloWorld();
  
  return 0;
}
